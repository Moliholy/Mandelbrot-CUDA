#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include <time.h>

int imageRows;
int imageColumns;
double imagePixelSize;
double centerPointReal;
double centerPointImaginary;
int maximumIterations;

void sequentialMandelbrot(int* matrix) {
        //initial coordinate (pixel (0, 0))
        double startReal = -((double) imageColumns * imagePixelSize) / 2.0
                        + centerPointReal;
        double startImaginary = -((double) imageRows * imagePixelSize) / 2.0
                        + centerPointImaginary;

        double transformation = 255.0 / (double) maximumIterations;

        //same as Wikipedia code
        int k;
        for (k = 0; k < imageRows * imageColumns; ++k) {
                int i = k / imageColumns;
                int j = k % imageColumns;

                double x0 = startReal + j * imagePixelSize;
                double y0 = startImaginary + i * imagePixelSize;
                double x = 0.0;
                double y = 0.0;
                int iteration = 0;

                while (x * x + y * y < 4.0 && iteration < maximumIterations) {
                        double xtemp = x * x - y * y + x0;
                        y = 2 * x * y + y0;
                        x = xtemp;
                        iteration++;
                }
                //changing the color
                matrix[i * imageColumns + j] = iteration * transformation;
        }
}

__global__
void parallelMandelbrot(int* matrix, int imageRows, int imageColumns,
                double imagePixelSize, double centerPointReal,
                double centerPointImaginary, int maximumIterations) {

        //same than wikipedia code
        int j = blockIdx.y * blockDim.y + threadIdx.y;
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        int pos = i * imageColumns + j;
        if (pos < imageRows * imageColumns) {

                //initial coordinate (pixel (0, 0))
                double startReal = -((double) imageColumns * imagePixelSize) / 2.0
                                + centerPointReal;
                double startImaginary = -((double) imageRows * imagePixelSize) / 2.0
                                + centerPointImaginary;

                double transformation = 255.0 / (double) maximumIterations;

                double x0 = startReal + j * imagePixelSize;
                double y0 = startImaginary + i * imagePixelSize;
                double x = 0.0;
                double y = 0.0;
                int iteration = 0;

                while (x * x + y * y < 4.0 && iteration < maximumIterations) {
                        double xtemp = x * x - y * y + x0;
                        y = 2 * x * y + y0;
                        x = xtemp;
                        iteration++;
                }
                //changing the color
                matrix[pos] = iteration * transformation;
        }
}

/**
 * Usage: ./mandel-magic imageRows imageColumns pixelSize centerPointReal
 *  centerPointImaginary maximumIterations
 */
int main(int argc, char *argv[]) {
        if (argc < 7) {
                printf("No enough arguments.");
                return -1;
        }

        imageRows = atoi(argv[1]);
        imageColumns = atoi(argv[2]);
        imagePixelSize = atof(argv[3]);
        centerPointReal = atof(argv[4]);
        centerPointImaginary = atof(argv[5]);
        maximumIterations = atoi(argv[6]);

        int* matrix = (int*) malloc(imageRows * imageColumns * sizeof(int));

        //sequential execution
        clock_t t = clock();
        sequentialMandelbrot(matrix);
        float sequentialExecutionTime = ((double) (clock() - t))
                        / ((double) (CLOCKS_PER_SEC));

        //starting parallel execution
        t = clock();
        int* d_matrix;

        //allocating memory
        hipMalloc(&d_matrix, imageRows * imageColumns * sizeof(int));

        //dimensions
        dim3 threadblock(16, 16);
        dim3 grid(1 + imageColumns / threadblock.x, 1 + imageRows / threadblock.y);

        //calling function
        parallelMandelbrot<<<grid, threadblock>>>(d_matrix, imageRows, imageColumns,
                        imagePixelSize, centerPointReal, centerPointImaginary,
                        maximumIterations);

        //synchronizing
        hipDeviceSynchronize();

        //once the function has been called I copy the result in matrix
        hipMemcpy(matrix, d_matrix, imageRows * imageColumns * sizeof(int),
                        hipMemcpyDeviceToHost);

        double parallelExecutionTime = ((double) (clock() - t))
                        / ((double) (CLOCKS_PER_SEC));

        printf("%d;%f;%f\n", imageRows, sequentialExecutionTime,
                        parallelExecutionTime);

        hipFree(d_matrix);
        free(matrix);

        return 0;
}
